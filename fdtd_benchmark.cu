#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// Include the header file
#include "maxwells.cuh" 

// Utility function to check CUDA errors
#define cudaCheckError() { \
    hipError_t e=hipGetLastError(); \
    if(e!=hipSuccess) { \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
        exit(EXIT_FAILURE); \
    } \
}

// Structure to hold benchmark results
struct BenchmarkResult {
    double totalTime;
    double kernelTime;
    double memoryBandwidth;
    double computeEfficiency;
};

// Function to measure kernel execution time and performance
BenchmarkResult measureKernelPerformance(int nx, int ny, float dx, float dy, float C0_p_dt, int source_position, float t,
                                         float* ez, float* dz, float* hx, float* hy, const float* er, const float* mh) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Warm-up run
    fdtd_update<<<(nx * ny + 255) / 256, 256>>>(nx, ny, dx, dy, C0_p_dt, source_position, t, ez, dz, hx, hy, er, mh);
    hipDeviceSynchronize();

    // Timed run
    hipEventRecord(start);
    fdtd_update<<<(nx * ny + 255) / 256, 256>>>(nx, ny, dx, dy, C0_p_dt, source_position, t, ez, dz, hx, hy, er, mh);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Calculate memory bandwidth
    size_t bytesRead = nx * ny * sizeof(float) * 5;  // ez, hx, hy, er, mh
    size_t bytesWritten = nx * ny * sizeof(float) * 3;  // ez, hx, hy
    double gigaBytesPerSecond = (bytesRead + bytesWritten) / (milliseconds * 1e-3) / 1e9;

    // Calculate compute efficiency
    int deviceId;
    hipGetDevice(&deviceId);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    
    double flops = nx * ny * 26;  // Approximate FLOPs per kernel execution
    double teraFlopsPerSecond = flops / (milliseconds * 1e-3) / 1e12;
    double peakTeraFlops = 2.0 * deviceProp.clockRate * deviceProp.multiProcessorCount * 32 / 1e6;  // Assuming 32 CUDA cores per SM
    double computeEfficiency = teraFlopsPerSecond / peakTeraFlops * 100.0;

    return {milliseconds / 1000.0, milliseconds / 1000.0, gigaBytesPerSecond, computeEfficiency};
}

// Function to run benchmarks
void runBenchmarks() {
    std::vector<std::pair<int, int>> gridSizes = {{100, 100}, {500, 500}, {1000, 1000}, {2000, 2000}};
    int steps = 1000;

    for (const auto& size : gridSizes) {
        int nx = size.first;
        int ny = size.second;
        int source_position = (nx / 2) * nx + (ny / 2);

        // Allocate memory on device
        float *ez, *dz, *hx, *hy, *er, *mh;
        hipMalloc(&ez, nx * ny * sizeof(float));
        hipMalloc(&dz, nx * ny * sizeof(float));
        hipMalloc(&hx, nx * ny * sizeof(float));
        hipMalloc(&hy, nx * ny * sizeof(float));
        hipMalloc(&er, nx * ny * sizeof(float));
        hipMalloc(&mh, nx * ny * sizeof(float));

        // Initialize fields on the device
        init_fields<<<(nx * ny + 255) / 256, 256>>>(nx, ny, ez, dz, hx, hy, er, mh);
        hipDeviceSynchronize();

        // Measure total execution time
        auto start = std::chrono::high_resolution_clock::now();

        BenchmarkResult totalResult = {0.0, 0.0, 0.0, 0.0};
        for (int step = 0; step < steps; ++step) {
            float t = step * dt;
            BenchmarkResult stepResult = measureKernelPerformance(nx, ny, dx, dy, C0_p_dt, source_position, t, ez, dz, hx, hy, er, mh);
            totalResult.kernelTime += stepResult.kernelTime;
            totalResult.memoryBandwidth += stepResult.memoryBandwidth;
            totalResult.computeEfficiency += stepResult.computeEfficiency;
        }

        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> diff = end - start;
        totalResult.totalTime = diff.count();

        // Calculate average metrics
        totalResult.memoryBandwidth /= steps;
        totalResult.computeEfficiency /= steps;

        // Print performance metrics
        std::cout << "Grid size: " << nx << "x" << ny << std::endl;
        std::cout << "Total time: " << totalResult.totalTime << " seconds" << std::endl;
        std::cout << "Total kernel time: " << totalResult.kernelTime << " seconds" << std::endl;
        std::cout << "Average memory bandwidth: " << totalResult.memoryBandwidth << " GB/s" << std::endl;
        std::cout << "Average compute efficiency: " << totalResult.computeEfficiency << "%" << std::endl;
        std::cout << std::endl;

        // Free allocated memory
        hipFree(ez);
        hipFree(dz);
        hipFree(hx);
        hipFree(hy);
        hipFree(er);
        hipFree(mh);
    }
}

int main() {
    runBenchmarks();
    return 0;
}