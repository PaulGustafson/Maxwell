#include <iostream>
#include <cmath>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>

constexpr float C0 = 299792458.0f; 

__device__ float update_curl_ex (int nx, int cell_x, int cell_y, int cell_id, float dy, const float * ez) {
  const int top_neighbor_id = nx * (cell_y + 1) + cell_x;
  return (ez[top_neighbor_id] - ez[cell_id]) / dy;
}

__device__ float update_curl_ey (
  int nx, int cell_x, int cell_y, int cell_id,
  float dx, const float * ez) {
  const int right_neighbor_id = cell_x == nx - 1 ? cell_y * nx + 0 : cell_id + 1;
  return -(ez[right_neighbor_id] - ez[cell_id]) / dx;
}

__device__ void update_h (
  int nx, int cell_id,
  float dx, float dy,
  const float *ez, const float *mh,
  float *hx, float *hy) {
  const int cell_x = cell_id % nx;
  const int cell_y = cell_id / nx;
  const float cex = update_curl_ex(nx, cell_x, cell_y, cell_id, dy, ez);
  const float cey = update_curl_ey(nx, cell_x, cell_y, cell_id, dx, ez);
  hx[cell_id] -= mh[cell_id] * cex;
  hy[cell_id] -= mh[cell_id] * cey;
}

__device__ static float update_curl_h (
  int nx, int cell_id, int cell_x, int cell_y, float dx, float dy,
  const float *hx, const float *hy) {
  const int left_neighbor_id = cell_x == 0 ? cell_y * nx + nx - 1 : cell_id - 1;
  const int bottom_neighbor_id = nx * (cell_y - 1) + cell_x;
  return (hy[cell_id] - hy[left_neighbor_id]) / dx
       - (hx[cell_id] - hx[bottom_neighbor_id]) / dy;
}

__device__ float gaussian_pulse (float t, float t_0, float tau) {
  return __expf (-(((t - t_0) / tau) * (t - t_0) / tau));
}

__device__ float calculate_source (float t, float frequency) {
  const float tau = 0.5f / frequency;
  const float t_0 = 6.0f * tau;
  return gaussian_pulse (t, t_0, tau);
}

__device__ void update_e (
  int nx, int cell_id, int own_in_process_begin, int source_position,
  float t, float dx, float dy, float C0_p_dt,
  float *ez, float *dz,
  const float *er,const float *hx, const float *hy) {
  const int cell_x = cell_id % nx;
  const int cell_y = cell_id / nx;
  const float chz = update_curl_h (nx, cell_id, cell_x, cell_y, dx, dy, hx, hy);
  dz[cell_id] += C0_p_dt * chz;
  if ((own_in_process_begin + cell_y) * nx + cell_x == source_position)
    dz[cell_id] += calculate_source (t, 5E+7);
  ez[cell_id] = dz[cell_id] / er[cell_id];
}

// constexpr int nx = 100;  // Grid size
// constexpr int ny = 100;
constexpr float dx = 1.0f;
constexpr float dy = 1.0f;
constexpr float dt = 1e-9;  // Time step
constexpr float C0_p_dt = C0 * dt;

// Device arrays
float *ez, *dz, *hx, *hy, *er, *mh;

// Kernel to initialize the arrays
__global__ void init_fields(int nx, int ny, float *ez, float *dz, float *hx, float *hy, float *er, float *mh) {
    int cell_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (cell_id < nx * ny) {
        ez[cell_id] = 0.0f;
        dz[cell_id] = 0.0f;
        hx[cell_id] = 0.0f;
        hy[cell_id] = 0.0f;
        er[cell_id] = 1.0f;  // Free space
        mh[cell_id] = 1.0f;
    }
}

// Kernel for running FDTD updates
__global__ void fdtd_update(int nx, int ny, float dx, float dy, float C0_p_dt, int source_position, float t,
                            float *ez, float *dz, float *hx, float *hy, const float *er, const float *mh) {
    int cell_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (cell_id < nx * ny) {
        update_h(nx, cell_id, dx, dy, ez, mh, hx, hy);
        update_e(nx, cell_id, 0, source_position, t, dx, dy, C0_p_dt, ez, dz, er, hx, hy);
    }
}

int main(int argc, char** argv) {
    if (argc < 5) {
        std::cerr << "Usage: " << argv[0] << " nx ny steps inc" << std::endl;
        return 1;
    }

    // Read grid size, steps, and increment from command line arguments
    int grid_size_x = 0, grid_size_y = 0, steps = 0, increment = 0;
    for (int i = 1; i < argc; i += 2) {
        if (std::string(argv[i]) == "--grid_size_x") grid_size_x = std::atoi(argv[i + 1]);
        else if (std::string(argv[i]) == "--grid_size_y") grid_size_y = std::atoi(argv[i + 1]);
        else if (std::string(argv[i]) == "--steps") steps = std::atoi(argv[i + 1]);
        else if (std::string(argv[i]) == "--increment") increment = std::atoi(argv[i + 1]);
    }

    // Assign to nx and ny for compatibility with existing code
    int nx = grid_size_x;
    int ny = grid_size_y;

    std::cout << "nx: " << nx << " ny: " << ny << " steps: " << steps << " increment: " << increment << std::endl;

    // Grid dimensions
    int source_position = (nx / 2) * nx + (ny / 2);  // Center of the grid
    
    // Allocate memory on device
    hipMalloc(&ez, nx * ny * sizeof(float));
    hipMalloc(&dz, nx * ny * sizeof(float));
    hipMalloc(&hx, nx * ny * sizeof(float));
    hipMalloc(&hy, nx * ny * sizeof(float));
    hipMalloc(&er, nx * ny * sizeof(float));
    hipMalloc(&mh, nx * ny * sizeof(float));

    // Initialize fields on the device
    init_fields<<<(nx * ny + 255) / 256, 256>>>(nx, ny, ez, dz, hx, hy, er, mh);
    hipDeviceSynchronize();

    // Time-stepping loop
    for (int step = 0; step < steps; ++step) {
        float t = step * dt;
        fdtd_update<<<(nx * ny + 255) / 256, 256>>>(nx, ny, dx, dy, C0_p_dt, source_position, t, ez, dz, hx, hy, er, mh);
        hipDeviceSynchronize();

        // Optionally print or log values of the field at the center of the grid
        if (step % increment == 0) {
            float ez_center;
            hipMemcpy(&ez_center, &ez[source_position], sizeof(float), hipMemcpyDeviceToHost);
            std::cout << "Step " << step << ", t = " << t << ", Ez at center: " << ez_center << std::endl;

            // Print to file
            float *h_ez = (float*)malloc(nx * ny * sizeof(float));  // Host-side copy of ez
            hipMemcpy(h_ez, ez, nx * ny * sizeof(float), hipMemcpyDeviceToHost);
            std::ofstream file("data/ez_step_" + std::to_string(step) + ".txt");
            for (int y = 0; y < ny; ++y) {
                for (int x = 0; x < nx; ++x) {
                    file << h_ez[y * nx + x] << " ";
                }
                file << "\n";
            }
            file.close();
        }
    }

    // Free memory
    hipFree(ez);
    hipFree(dz);
    hipFree(hx);
    hipFree(hy);
    hipFree(er);
    hipFree(mh);

    return 0;
}
