#include <iostream>
#include <cmath>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>

constexpr float C0 = 299792458.0f; 
constexpr float dx = 1.0f;
constexpr float dy = 1.0f;
constexpr float dt = 1e-9;
constexpr float C0_p_dt = C0 * dt;

__device__ float update_curl_ex (int nx, int cell_x, int cell_y, int cell_id, float dy, const float * ez) {
    const int top_neighbor_id = nx * (cell_y + 1) + cell_x;
    return (ez[top_neighbor_id] - ez[cell_id]) / dy;
}

__device__ float update_curl_ey (int nx, int cell_x, int cell_y, int cell_id, float dx, const float * ez) {
    const int right_neighbor_id = cell_x == nx - 1 ? cell_y * nx + 0 : cell_id + 1;
    return -(ez[right_neighbor_id] - ez[cell_id]) / dx;
}

__device__ void update_h (int nx, int cell_id, float dx, float dy, const float *ez, const float *mh, float *hx, float *hy) {
    const int cell_x = cell_id % nx;
    const int cell_y = cell_id / nx;
    const float cex = update_curl_ex(nx, cell_x, cell_y, cell_id, dy, ez);
    const float cey = update_curl_ey(nx, cell_x, cell_y, cell_id, dx, ez);
    hx[cell_id] -= mh[cell_id] * cex;
    hy[cell_id] -= mh[cell_id] * cey;
}

__device__ float update_curl_h (int nx, int cell_id, int cell_x, int cell_y, float dx, float dy, const float *hx, const float *hy) {
    const int left_neighbor_id = cell_x == 0 ? cell_y * nx + nx - 1 : cell_id - 1;
    const int bottom_neighbor_id = nx * (cell_y - 1) + cell_x;
    return (hy[cell_id] - hy[left_neighbor_id]) / dx - (hx[cell_id] - hx[bottom_neighbor_id]) / dy;
}

__device__ float gaussian_pulse (float t, float t_0, float tau) {
    return __expf(-(((t - t_0) / tau) * (t - t_0) / tau));
}

__device__ float calculate_source(float t, float frequency) {
    const float tau = 0.5f / frequency;
    const float t_0 = 6.0f * tau;
    return gaussian_pulse(t, t_0, tau);
}

__device__ void update_e(int nx, int cell_id, int own_in_process_begin, int source_position, float t, float dx, float dy, float C0_p_dt, float *ez, float *dz, const float *er, const float *hx, const float *hy) {
    const int cell_x = cell_id % nx;
    const int cell_y = cell_id / nx;
    const float chz = update_curl_h(nx, cell_id, cell_x, cell_y, dx, dy, hx, hy);
    dz[cell_id] += C0_p_dt * chz;
    if ((own_in_process_begin + cell_y) * nx + cell_x == source_position)
        dz[cell_id] += calculate_source(t, 5E+7);
    ez[cell_id] = dz[cell_id] / er[cell_id];
}

__global__ void init_fields(int nx, int ny, float *ez, float *dz, float *hx, float *hy, float *er, float *mh) {
    int cell_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (cell_id < nx * ny) {
        ez[cell_id] = 0.0f;
        dz[cell_id] = 0.0f;
        hx[cell_id] = 0.0f;
        hy[cell_id] = 0.0f;
        er[cell_id] = 1.0f;
        mh[cell_id] = 1.0f;
    }
}


void check_cuda_error(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(result) << std::endl;
        exit(1);
    }
}

__global__ void update_h_kernel(int nx, int ny, float dx, float dy,
  const float *ez, const float *mh,
  float *hx, float *hy, int total, int min_y, int max_y) {

  const int cell_id = blockIdx.x * blockDim.x + threadIdx.x;
  if ((cell_id < total) ){
    const int cell_x = cell_id % nx;
    const int cell_y = cell_id / nx;
    if( (cell_y >=  min_y) && (cell_y < max_y) ) {
        //
        const float cex = update_curl_ex(nx, cell_x, cell_y, cell_id, dy, ez);
        const float cey = update_curl_ey(nx, cell_x, cell_y, cell_id, dx, ez);

        //! Save 1 global memory call
        const float mh_id = mh[cell_id];
        hx[cell_id] -= mh_id * cex;
        hy[cell_id] -= mh_id * cey;
    }
  }
}

__global__ void update_e_kernel(int nx, int source_position,
    float t, float dx, float dy, float C0_p_dt,
    float *ez, float *dz,
    const float *er,const float *hx, const float *hy, int total, int min_y, int max_y) {

    const int cell_id = blockIdx.x * blockDim.x + threadIdx.x;

    if ((cell_id < total) ){  
        const int cell_x = cell_id % nx;
        const int cell_y = cell_id / nx;
        if( (cell_y >=  min_y) && (cell_y < max_y)) {

            const float chz = update_curl_h (nx, cell_id, cell_x, cell_y, dx, dy, hx, hy);
            dz[cell_id] += C0_p_dt * chz;
            if ((cell_y) * nx + cell_x == source_position)
                dz[cell_id] += calculate_source (t, 5E+7);
            ez[cell_id] = dz[cell_id] / er[cell_id];
            
        }
    }
}

int main(int argc, char** argv) {
    if (argc < 5) {
        std::cerr << "Usage: " << argv[0] << " nx ny steps inc" << std::endl;
        return 1;
    }

    int nx = 0, ny = 0, steps = 0, increment = 0;
    for (int i = 1; i < argc; i += 2) {
        if (std::string(argv[i]) == "--grid_size_x") nx = std::atoi(argv[i + 1]);
        else if (std::string(argv[i]) == "--grid_size_y") ny = std::atoi(argv[i + 1]);
        else if (std::string(argv[i]) == "--steps") steps = std::atoi(argv[i + 1]);
        else if (std::string(argv[i]) == "--increment") increment = std::atoi(argv[i + 1]);
    }

    // Check if all required parameters were provided
    if (nx == 0 || ny == 0 || steps == 0 || increment == 0) {
        std::cerr << "Error: Missing or invalid command line arguments" << std::endl;
        std::cerr << "Usage: " << argv[0] << " --grid_size_x <nx> --grid_size_y <ny> --steps <steps> --increment <increment>" << std::endl;
        return 1;
    }
    // Print out the args for debugging
    std::cout << "Debugging: Command line arguments" << std::endl;
    std::cout << "grid_size_x (nx): " << nx << std::endl;
    std::cout << "grid_size_y (ny): " << ny << std::endl;
    std::cout << "steps: " << steps << std::endl;
    std::cout << "increment: " << increment << std::endl;
    std::cout << std::endl;

    int source_position = (nx / 2) * nx + (ny / 2);

    int num_devices = 2;
    check_cuda_error(hipSetDevice(0), "Unable to set device 0");
    float *ez_0, *dz_0, *hx_0, *hy_0, *er_0, *mh_0;
    check_cuda_error(hipMalloc(&ez_0, (nx * ny) * sizeof(float)), "Allocating ez_0");
    check_cuda_error(hipMalloc(&dz_0, (nx * ny) * sizeof(float)), "Allocating dz_0");
    check_cuda_error(hipMalloc(&hx_0, (nx * ny) * sizeof(float)), "Allocating hx_0");
    check_cuda_error(hipMalloc(&hy_0, (nx * ny) * sizeof(float)), "Allocating hy_0");
    check_cuda_error(hipMalloc(&er_0, (nx * ny) * sizeof(float)), "Allocating er_0");
    check_cuda_error(hipMalloc(&mh_0, (nx * ny) * sizeof(float)), "Allocating mh_0");

    check_cuda_error(hipSetDevice(1), "Unable to set device 1");
    float *ez_1, *dz_1, *hx_1, *hy_1, *er_1, *mh_1;
    check_cuda_error(hipMalloc(&ez_1, (nx * ny) * sizeof(float)), "Allocating ez_1");
    check_cuda_error(hipMalloc(&dz_1, (nx * ny) * sizeof(float)), "Allocating dz_1");
    check_cuda_error(hipMalloc(&hx_1, (nx * ny) * sizeof(float)), "Allocating hx_1");
    check_cuda_error(hipMalloc(&hy_1, (nx * ny) * sizeof(float)), "Allocating hy_1");
    check_cuda_error(hipMalloc(&er_1, (nx * ny) * sizeof(float)), "Allocating er_1");
    check_cuda_error(hipMalloc(&mh_1, (nx * ny) * sizeof(float)), "Allocating mh_1");

    check_cuda_error(hipSetDevice(0), "Unable to set device 0");
    init_fields<<<(nx * ny + 255) / 256, 256>>>(nx, ny, ez_0, dz_0, hx_0, hy_0, er_0, mh_0);
    hipDeviceSynchronize();

    check_cuda_error(hipSetDevice(1), "Unable to set device 1");
    init_fields<<<(nx * ny + 255) / 256, 256>>>(nx, ny, ez_1, dz_1, hx_1, hy_1, er_1, mh_1);
    hipDeviceSynchronize();

    // Time-stepping loop
    for (int step = 0; step < steps; ++step) {
        float t = step * dt;

        // GPU 0 update
        check_cuda_error(hipSetDevice(0), "Unable to set device 0");
        int total = nx * ny ;
        update_e_kernel<<<(nx * ny  + 255) / 256, 256>>>(nx, source_position, t, dx,dy, C0_p_dt, ez_0, dz_0, er_0, hx_0, hy_0,  total, 0, nx/2 + 1);
        
        
        // GPU 1 update
        check_cuda_error(hipSetDevice(1), "Unable to set device 1");
        update_h_kernel<<<(nx * ny  + 255) / 256, 256>>>(nx, ny, dx, dy, ez_1, mh_1, hx_1, hy_1, total, nx/2 - 1, nx);
        
        

        // Synchronize updates
        hipDeviceSynchronize();

        // // Exchange boundary data (between GPU 0 and GPU 1)
        // // Copy the last row from GPU 0 to GPU 1 and the first row from GPU 1 to GPU 0
        float *boundary_row_0 = (float*)malloc(nx * sizeof(float));
        float *boundary_row_1 = (float*)malloc(nx * sizeof(float));
        
        // Copy last row of ez_0 (the boundary) to host
        check_cuda_error(hipMemcpy(boundary_row_0, &ez_0[(ny / 2) * nx], nx * sizeof(float), hipMemcpyDeviceToHost), "Copy boundary row 0 to host");
        
        // Copy first row of ez_1 (the boundary) to host
        check_cuda_error(hipSetDevice(1), "Unable to set device 1");
        check_cuda_error(hipMemcpy(boundary_row_1, &ez_1[(ny / 2) * nx], nx * sizeof(float), hipMemcpyDeviceToHost), "Copy boundary row 1 to host");


        // Copy to GPU 1
        check_cuda_error(hipSetDevice(1), "Unable to set device 1");
        check_cuda_error(hipMemcpy(&ez_1[(ny / 2) * nx], boundary_row_0, nx * sizeof(float), hipMemcpyHostToDevice), "Copy boundary row 0 to GPU 1");

        // Copy to GPU 0
        check_cuda_error(hipSetDevice(0), "Unable to set device 0");
        check_cuda_error(hipMemcpy(&ez_0[(ny / 2) * nx], boundary_row_1, nx * sizeof(float), hipMemcpyHostToDevice), "Copy boundary row 1 to GPU 0");

        free(boundary_row_0);
        free(boundary_row_1);

        hipDeviceSynchronize();

        check_cuda_error(hipSetDevice(0), "Unable to set device 0");
        update_h_kernel<<<(nx * ny  + 255) / 256, 256>>>(nx, ny, dx, dy, ez_0, mh_0, hx_0, hy_0,  total, 0, nx/2 + 1);
        check_cuda_error(hipSetDevice(1), "Unable to set device 1");
        update_e_kernel<<<(nx * ny + 255) / 256, 256>>>(nx, source_position, t, dx,dy, C0_p_dt, ez_1, dz_1, er_1, hx_1, hy_1, total, nx/2 -1, nx);

        hipDeviceSynchronize();


        // Optionally print or log values of the field at the center of the grid
        if (step % increment == 0) {
            float ez_center_0, ez_center_1;
            check_cuda_error(hipMemcpy(&ez_center_0, &ez_0[source_position], sizeof(float), hipMemcpyDeviceToHost), "Copy ez_center_0");
            check_cuda_error(hipMemcpy(&ez_center_1, &ez_1[source_position ], sizeof(float), hipMemcpyDeviceToHost), "Copy ez_center_1");
            std::cout << "Step " << step << ", t = " << t << ", Ez at center GPU 0: " << ez_center_0 << ", Ez at center GPU 1: " << ez_center_1 << std::endl;

            float *h_ez_0 = (float*)malloc(nx * ny * sizeof(float));  // Host-side copy for GPU 0
            float *h_ez_1 = (float*)malloc(nx * ny * sizeof(float));  // Host-side copy for GPU 1

            // Copy data from both GPUs
            check_cuda_error(hipMemcpy(h_ez_0, ez_0, nx * ny * sizeof(float), hipMemcpyDeviceToHost), "Copy ez_0 to host");
            check_cuda_error(hipMemcpy(h_ez_1, ez_1, nx * ny * sizeof(float), hipMemcpyDeviceToHost), "Copy ez_1 to host");

            // Print to file
            std::ofstream file("data/ez_step_" + std::to_string(step) + ".txt");
            if (file.is_open()) {
                // Write data from GPU 0
                for (int y = 0; y < ny / 2; ++y) {
                    for (int x = 0; x < nx; ++x) {
                        file << h_ez_0[y * nx + x] << " ";
                    }
                    file << "\n";
                }
                
                // Write data from GPU 1
                for (int y = ny / 2; y < ny ; ++y) {
                    for (int x = 0; x < nx; ++x) {
                        file << h_ez_1[y * nx + x] << " ";
                    }
                    file << "\n";
                }
                
                file.close();
            } else {
                std::cerr << "Unable to open file for writing" << std::endl;
            }

            // Free host memory
            free(h_ez_0);
            free(h_ez_1);
        }

    }

    // Free memory
    hipFree(ez_0);
    hipFree(ez_1);
    hipFree(dz_0);
    hipFree(dz_1);
    hipFree(hx_0);
    hipFree(hx_1);
    hipFree(hy_0);
    hipFree(hy_1);
    hipFree(er_0);
    hipFree(er_1);
    hipFree(mh_0);
    hipFree(mh_1);

    return 0;
}