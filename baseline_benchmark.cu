
#include <iostream>
#include <cmath>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include "baseline_single_kernel.hh"

int main() {

    int nx = 1024;
    int ny = 1024;
    int n_cells = nx * ny;
    int n_steps = 5;

    // Allocate memory on device
    // Device arrays
    float *ez, *dz, *hx, *hy, *er, *mh;
    hipMalloc(&ez, n_cells * sizeof(float));
    hipMalloc(&dz, n_cells * sizeof(float));
    hipMalloc(&hx, n_cells * sizeof(float));
    hipMalloc(&hy, n_cells * sizeof(float));
    hipMalloc(&er, n_cells * sizeof(float));
    hipMalloc(&mh, n_cells * sizeof(float));

    // warm up
    {
        NvtxScope scope("warmup");
        baseline_fdtd_cg(
        n_steps,
        nx, ny, n_cells, 
        ez, 
        mh,
        hx, hy,
        dz,
        er);
    }

    {
        NvtxScope scope("warmup");
        // actual running
        baseline_fdtd_cg(
        n_steps,
        nx, ny, n_cells, 
        ez, 
        mh,
        hx, hy,
        dz,
        er);
    }

    // Free memory
    hipFree(ez);
    hipFree(dz);
    hipFree(hx);
    hipFree(hy);
    hipFree(er);
    hipFree(mh);

    return 0;
}